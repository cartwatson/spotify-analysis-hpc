#include "hip/hip_runtime.h"
#include <chrono>
#include <fstream>
#include <iostream>
#include <sstream>
#include <vector>
#include <random>
#include <assert.h>

#include "util.cpp"

#define BLOCKSIZE 256


struct Song {
    float feature1, feature2, feature3;
    int cluster;

    Song(): feature1(0.0), feature2(0.0), feature3(0.0), cluster(-1) {}

    Song(float f1, float f2, float f3):
        feature1(f1),
        feature2(f2),
        feature3(f3),
        cluster(-1)
    {}
};

struct Centroid {
    float feature1, feature2, feature3;
    int cluster_size;

    Centroid(): feature1(0.0), feature2(0.0), feature3(0.0), cluster_size(0) {}

    Centroid(float f1, float f2, float f3):
        feature1(f1),
        feature2(f2),
        feature3(f3),
        cluster_size(0)
    {}
};

/**
 * Calculates the distance between two points in 3D space (no need to get the square root, it's all relative)
*/
__device__ double sq_distance(Song* s1, Centroid* c)
{
    return (s1->feature1 - c->feature1) * (s1->feature1 - c->feature1) +
           (s1->feature2 - c->feature2) * (s1->feature2 - c->feature2) +
           (s1->feature3 - c->feature3) * (s1->feature3 - c->feature3);
}

/**
 * Assigns each song to the cluster with the closest centroid
*/
__global__ void assignSongToCluster(Song* songs, Centroid* centroids, int n, int k)
{
    extern __shared__ Centroid shared_centroids[];
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadIdx.x < k)
        shared_centroids[threadIdx.x] = centroids[threadIdx.x];
    __syncthreads();

    if (gid < n)
    {
        double min_dist = sq_distance(&songs[gid], &shared_centroids[0]);
        int cluster = 0;
        for (int c = 1; c < k; ++c)
        {
            double dist = sq_distance(&songs[gid], &shared_centroids[c]);
            if (dist < min_dist)
            {
                min_dist = dist;
                cluster = c;
            }
        }
        songs[gid].cluster = cluster;
    }
}

__global__ void calculateNewCentroids(Song* songs, Centroid* centroids, int n)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < n)
    {
        int cluster = songs[gid].cluster; // Get the cluster of each song
        atomicAdd(&centroids[cluster].feature1, songs[gid].feature1);
        atomicAdd(&centroids[cluster].feature2, songs[gid].feature2);
        atomicAdd(&centroids[cluster].feature3, songs[gid].feature3);
        atomicAdd(&centroids[cluster].cluster_size, 1);
    }
}

inline hipError_t checkCuda(hipError_t result)
{
    if (result != hipSuccess) {
        std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
        assert(result == hipSuccess);
    }
    return result;
}

void kMeansCUDA(Song* songs, int n, int epochs, int k)
{
    Song* songs_d;
    checkCuda(hipMalloc(&songs_d, n * sizeof(Song)));
    checkCuda(hipMemcpy(songs_d, songs, n * sizeof(Song), hipMemcpyHostToDevice));

    #ifdef TESTING
    std::mt19937 rng(123);
    #else
    std::mt19937 rng(static_cast<unsigned>(std::time(0)));
    #endif
    Centroid* centroids = new Centroid[k];
    Centroid* centroids_d;
    for (int i = 0; i < k; ++i)
    {
        int rand_idx = rng() % n;
        centroids[i] = Centroid(songs[rand_idx].feature1, songs[rand_idx].feature2, songs[rand_idx].feature3);
    }
    checkCuda(hipMalloc(&centroids_d, k*sizeof(Centroid)));
    checkCuda(hipMemcpy(centroids_d, centroids, k*sizeof(Centroid), hipMemcpyHostToDevice));

    int nBlocks = (n + BLOCKSIZE - 1) / BLOCKSIZE;
    dim3 gridDim(nBlocks, 1, 1);
    dim3 blockDim(BLOCKSIZE, 1, 1);

    for (int epoch = 0; epoch < epochs; ++epoch)
    {
        assignSongToCluster<<<gridDim, blockDim>>>(songs_d, centroids_d, n, k); // Use the centroids to assign each song to a cluster
        checkCuda(hipGetLastError());
        checkCuda(hipDeviceSynchronize());

        checkCuda(hipMemset(centroids_d, 0, k*sizeof(Centroid)));

        calculateNewCentroids<<<gridDim, blockDim>>>(songs_d, centroids_d, n);
        checkCuda(hipGetLastError());
        checkCuda(hipDeviceSynchronize());

        checkCuda(hipMemcpy(centroids, centroids_d, k*sizeof(Centroid), hipMemcpyDeviceToHost));
        for (int i = 0; i < k; ++i)
        {
            centroids[i].feature1 /= centroids[i].cluster_size;
            centroids[i].feature2 /= centroids[i].cluster_size;
            centroids[i].feature3 /= centroids[i].cluster_size;
        }
        checkCuda(hipMemcpy(centroids_d, centroids, k*sizeof(Centroid), hipMemcpyHostToDevice));
    }

    checkCuda(hipMemcpy(songs, songs_d, n*sizeof(Song), hipMemcpyDeviceToHost));
}


int main(int argc, char* argv[])
{
    int maxLines = 250000;
    if (argc > 1)
    {
        maxLines = std::stoi(argv[1]);
        if (maxLines < 0 || maxLines > MAX_LINES)
            maxLines = MAX_LINES;
        std::cout << "maxLines = " << maxLines << std::endl;
    }

    auto start = std::chrono::high_resolution_clock::now();
    
    std::vector<double*> data = parseCSV(maxLines);
    std::vector<std::string> featureNames = {"danceability", "acousticness", "liveness"};

    Song* songs = new Song[data.size()];
    for (size_t i = 0; i < data.size(); ++i)
        songs[i] = Song(data[i][0], data[i][6], data[i][8]);

    auto endParse = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = endParse - start;
    std::cout << "Parsed data in " << duration.count() << " seconds" << std::endl;
    
    std::cout << "Running k-means..." << std::endl;

    kMeansCUDA(songs, data.size(), 100, 5);

    auto endkMeans = std::chrono::high_resolution_clock::now();
    duration = endkMeans - endParse;
    std::cout << "Finished k-means in " << duration.count() << " seconds" << std::endl;

    std::cout << "Writing output to file..." << std::endl;
    std::string header = featureNames[0] + "," + featureNames[1] + "," + featureNames[2] + ",cluster";

    std::vector<double*> output;
    for (size_t i = 0; i < data.size(); ++i)
    {
        double* row = new double[4];
        row[0] = songs[i].feature1;
        row[1] = songs[i].feature2;
        row[2] = songs[i].feature3;
        row[3] = songs[i].cluster;
        output.push_back(row);
    }

    writeCSV(output, "src/data/output.csv", header);

    return 0;
}