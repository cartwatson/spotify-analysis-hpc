#include "hip/hip_runtime.h"
#include <chrono>
#include <fstream>
#include <iostream>
#include <sstream>
#include <vector>
#include <random>
#include <assert.h>

#include "util.cpp"

#define FEATURES 3
#define BLOCKSIZE 256
#define EPOCHS 2
#define K 5

inline hipError_t checkCuda(hipError_t result)
{
    if (result != hipSuccess) {
        std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
        assert(result == hipSuccess);
    }
    return result;
}

/**
 * Calculates the distance between two points in 3D space (no need to get the square root, it's all relative)
*/
__device__ double sq_distance(float f1, float f2, float f3, float c1, float c2, float c3)
{
    return (f1 - c1) * (f1 - c1) +
           (f2 - c2) * (f2 - c2) +
           (f3 - c3) * (f3 - c3);
}

__global__ void epochIter(float* songs, int* clusterAssignments, float* centroids, int* clusterCounts, int n)
{
    __shared__ float s_centroids[K*FEATURES];
    __shared__ int s_clusterCounts[K];

    __shared__ float s_songs[BLOCKSIZE*FEATURES];
    __shared__ int s_clusterAssignments[BLOCKSIZE];

    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= n) return;

    // Load centroids into shared memory
    int tid = threadIdx.x;
    if (tid < K)
    {
        s_centroids[tid*FEATURES] = centroids[tid*FEATURES];
        s_centroids[tid*FEATURES+1] = centroids[tid*FEATURES+1];
        s_centroids[tid*FEATURES+2] = centroids[tid*FEATURES+2];
        s_clusterCounts[tid] = 0;
    }

    // Load songs into shared memory
    s_songs[tid*FEATURES] = songs[gid*FEATURES];
    s_songs[tid*FEATURES+1] = songs[gid*FEATURES+1];
    s_songs[tid*FEATURES+2] = songs[gid*FEATURES+2];
    s_clusterAssignments[tid] = -1;

    __syncthreads();

    // Find closest centroid for each song
    double minDist = sq_distance(s_songs[tid*FEATURES], s_songs[tid*FEATURES+1], s_songs[tid*FEATURES+2],
                                    s_centroids[0], s_centroids[1], s_centroids[2]);
    int closestClust = 0;
    for (int i = 1; i < K; ++i)
    {
        double dist = sq_distance(s_songs[tid*FEATURES], s_songs[tid*FEATURES+1], s_songs[tid*FEATURES+2],
                                    s_centroids[i*FEATURES], s_centroids[i*FEATURES+1], s_centroids[i*FEATURES+2]);
        if (dist < minDist)
        {
            minDist = dist;
            closestClust = i;
        }
    }
    s_clusterAssignments[tid] = closestClust;

    __syncthreads();

    // Update cluster counts
    atomicAdd(&s_clusterCounts[s_clusterAssignments[tid]], 1);

    __syncthreads();

    // Update centroids
    if (tid < K)
    {
        int count = s_clusterCounts[tid];
        if (count > 0)
        {
            float newCentroid[FEATURES] = {0};
            for (int i = 0; i < BLOCKSIZE; ++i)
            {
                if (s_clusterAssignments[i] == tid)
                {
                    newCentroid[0] += s_songs[i*FEATURES];
                    newCentroid[1] += s_songs[i*FEATURES+1];
                    newCentroid[2] += s_songs[i*FEATURES+2];
                }
            }
            s_centroids[tid*FEATURES] = newCentroid[0] / count;
            s_centroids[tid*FEATURES+1] = newCentroid[1] / count;
            s_centroids[tid*FEATURES+2] = newCentroid[2] / count;
        }
    }

    __syncthreads();

    // Write back to global memory
    if (tid < K)
    {
        centroids[tid*FEATURES] = s_centroids[tid*FEATURES];
        centroids[tid*FEATURES+1] = s_centroids[tid*FEATURES+1];
        centroids[tid*FEATURES+2] = s_centroids[tid*FEATURES+2];
        clusterCounts[tid] = s_clusterCounts[tid];
    }
    if (tid < BLOCKSIZE)
        clusterAssignments[gid] = s_clusterAssignments[tid];
}

void kMeansCUDA(float* songs_h, int n)
{
    int allSongsSize = n*FEATURES*sizeof(float); // Song list size in bytes
    int allCentroidsSize = K*FEATURES*sizeof(float); // Centroid list size in bytes

    // Initialize songs on device
    float* songs_d;
    checkCuda(hipMalloc(&songs_d, allSongsSize));
    checkCuda(hipMemcpy(songs_d, songs_h, allSongsSize, hipMemcpyHostToDevice));

    std::mt19937 rng(123);
    float centroids[K*FEATURES];
    for (int i = 0; i < K; ++i)
    {
        int randIdx = rng() % n;
        memcpy(&centroids[i*FEATURES], &songs_h[randIdx*FEATURES], FEATURES*sizeof(float));
    }

    // Initialize centroids on device
    float* centroids_d;
    checkCuda(hipMalloc(&centroids_d, allCentroidsSize));
    checkCuda(hipMemcpy(centroids_d, centroids, allCentroidsSize, hipMemcpyHostToDevice));

    int nBlocks = (n + BLOCKSIZE - 1) / BLOCKSIZE;
    dim3 gridDim(nBlocks, 1, 1);
    dim3 blockDim(BLOCKSIZE, 1, 1);

    // initialize all cluster counts to 0 and cluster assignments to -1
    int* clusterAssignments = new int[n];
    for (int i = 0; i < n; ++i)
        clusterAssignments[i] = -1;

    int clusterCounts[K];
    for (int i = 0; i < K; ++i)
        clusterCounts[i] = 0;

    for (int epoch = 0; epoch < EPOCHS; ++epoch)
    {
        epochIter<<<gridDim, blockDim>>>(songs_d, clusterAssignments, centroids_d, clusterCounts, n);
        checkCuda(hipGetLastError());
        checkCuda(hipDeviceSynchronize());
    }
    checkCuda(hipMemcpy(songs_h, songs_d, allSongsSize, hipMemcpyDeviceToHost));
}


int main(int argc, char* argv[])
{
    int maxLines = 250000;
    if (argc > 1)
    {
        maxLines = std::stoi(argv[1]);
        if (maxLines < 0 || maxLines > MAX_LINES)
            maxLines = MAX_LINES;
        std::cout << "maxLines = " << maxLines << std::endl;
    }

    auto start = std::chrono::high_resolution_clock::now();
    
    std::vector<double*> allData = parseCSV(maxLines);
    std::vector<std::string> featureNames = {"danceability", "acousticness", "liveness"};

    float* songs = new float[allData.size()*FEATURES]; // +1 to leave room for cluster
    for (size_t i = 0; i < allData.size(); ++i)
    {
        songs[i*FEATURES] = allData[i][0];
        songs[i*FEATURES+1] = allData[i][6];
        songs[i*FEATURES+2] = allData[i][8];
    }

    auto endParse = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = endParse - start;
    std::cout << "Parsed data in " << duration.count() << " seconds" << std::endl;
    
    std::cout << "Running k-means..." << std::endl;

    kMeansCUDA(songs, allData.size());

    auto endkMeans = std::chrono::high_resolution_clock::now();
    duration = endkMeans - endParse;
    std::cout << "Finished k-means in " << duration.count() << " seconds" << std::endl;

    std::cout << "Writing output to file..." << std::endl;
    std::string header = featureNames[0] + "," + featureNames[1] + "," + featureNames[2] + ",cluster";

    std::vector<double*> output;
    for (size_t i = 0; i < allData.size(); ++i)
    {
        double* row = new double[4];
        row[0] = songs[i*FEATURES];
        row[1] = songs[i*FEATURES+1];
        row[2] = songs[i*FEATURES+2];
        row[3] = songs[i*FEATURES+3];
        output.push_back(row);
    }

    writeCSV(output, "src/data/output.csv", header);

    return 0;
}