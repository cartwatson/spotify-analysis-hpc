#include "hip/hip_runtime.h"
// Include Directives
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>
#include <random>
#include <sstream>
#include <fstream>
#include <assert.h>

#include "util.cpp"

// CUDA-specific code
struct Song {
    float feature1, feature2, feature3;
    int cluster;

    Song() : feature1(0.0), feature2(0.0), feature3(0.0), cluster(-1) {}

    Song(float f1, float f2, float f3) :
        feature1(f1),
        feature2(f2),
        feature3(f3),
        cluster(-1)
    {}
};

struct Centroid {
    float feature1, feature2, feature3;
    int cluster_size;

    Centroid() : feature1(0.0), feature2(0.0), feature3(0.0), cluster_size(0) {}

    Centroid(float f1, float f2, float f3) :
        feature1(f1),
        feature2(f2),
        feature3(f3),
        cluster_size(0)
    {}
};
inline hipError_t checkCuda(hipError_t result)
{
    if (result != hipSuccess) {
        std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
        assert(result == hipSuccess);
    }
    return result;
}
__device__ double sq_distance(Song* s1, Centroid* c)
{
    return (s1->feature1 - c->feature1) * (s1->feature1 - c->feature1) +
        (s1->feature2 - c->feature2) * (s1->feature2 - c->feature2) +
        (s1->feature3 - c->feature3) * (s1->feature3 - c->feature3);
}

__global__ void assignSongToCluster(Song* songs, Centroid* centroids, int n, int k)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    float minDist = 100000000;
    int cluster = -1;
    if (gid < n)
        for (int c = 0; c < k; ++c)
        {
            double dist = sq_distance(&songs[gid], &centroids[c]);
            if (dist < minDist)
            {
                minDist = dist;
                cluster = c;
            }
        }
    songs[gid].cluster = cluster;
}

__global__ void calculateNewCentroids(Song* songs, Centroid* centroids, int n)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < n)
    {
        int cluster = songs[gid].cluster; // Get the cluster of each song
        atomicAdd(&centroids[cluster].feature1, songs[gid].feature1);
        atomicAdd(&centroids[cluster].feature2, songs[gid].feature2);
        atomicAdd(&centroids[cluster].feature3, songs[gid].feature3);
        atomicAdd(&centroids[cluster].cluster_size, 1);
    }
}

// Wrapper function for assignSongToCluster kernel

extern "C" void callAssignSongToCluster(Song * songs, Centroid * centroids, int n, int k) {
    // Example: Assuming you want to use 256 threads per block
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock; // Ensure there are enough blocks

    dim3 grid(blocksPerGrid);
    dim3 block(threadsPerBlock);

    // Launch the kernel
    assignSongToCluster << <grid, block >> > (songs, centroids, n, k);

    // Check for errors and synchronize
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        // Optionally, throw an exception or perform other error handling
    }
}

// Wrapper function for calculateNewCentroids kernel
extern "C" void callCalculateNewCentroids(Song * songs, Centroid * centroids, int n) {
    // Example: Assuming you want to use 256 threads per block
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock; // Ensure there are enough blocks

    dim3 grid(blocksPerGrid);
    dim3 block(threadsPerBlock);

    // Launch the kernel
    calculateNewCentroids << <grid, block >> > (songs, centroids, n);

    // Check for errors and synchronize
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        // Optionally, throw an exception or perform other error handling
    }
}

// Definitions of CUDA wrapper functions
extern "C" void allocateMemoryAndCopyToGPU(Song** deviceSongs, Centroid** deviceCentroids, const Song* hostSongs, const Centroid* hostCentroids, int numSongs, int numCentroids) {
    // Allocate memory for songs on the device
    checkCuda(hipMalloc(deviceSongs, numSongs * sizeof(Song)));
    checkCuda(hipMemcpy(*deviceSongs, hostSongs, numSongs * sizeof(Song), hipMemcpyHostToDevice));

    // Allocate memory for centroids on the device
    checkCuda(hipMalloc(deviceCentroids, numCentroids * sizeof(Centroid)));
    checkCuda(hipMemcpy(*deviceCentroids, hostCentroids, numCentroids * sizeof(Centroid), hipMemcpyHostToDevice));
    // The code replaced in mainProgram:
    // checkCuda(hipMalloc(&localSongs_d, localN * sizeof(Song)));
    // checkCuda(hipMemcpy(localSongs_d, localSongs, localN * sizeof(Song), hipMemcpyHostToDevice));
    // checkCuda(hipMalloc(&centroids_d, k * sizeof(Centroid)));
    // checkCuda(hipMemcpy(centroids_d, centroids, k * sizeof(Centroid), hipMemcpyHostToDevice));
}

extern "C" void freeGPUMemory(Song* deviceSongs, Centroid* deviceCentroids) {
    checkCuda(hipFree(deviceSongs));
    checkCuda(hipFree(deviceCentroids));
    // Replaced code
    // checkCuda(hipFree(localSongs_d));
    // checkCuda(hipFree(centroids_d));
}

extern "C" void gpuErrorCheck() {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        assert(error == hipSuccess);
    }
    checkCuda(hipDeviceSynchronize());
    // // Replaced
    // checkCuda(hipGetLastError());
    // checkCuda(hipDeviceSynchronize());

}

extern "C" void resetCentroids(Centroid* centroids_d, int k) {
    checkCuda(hipMemset(centroids_d, 0, k * sizeof(Centroid)));
    // Replaced
    // checkCuda(hipMemset(centroids_d, 0, k * sizeof(Centroid)));
}

extern "C" void copyCentroidsToHost(Centroid* centroids, Centroid* centroids_d, int k) {
    checkCuda(hipMemcpy(centroids, centroids_d, k * sizeof(Centroid), hipMemcpyDeviceToHost));
    // replaced
     //   checkCuda(hipMemcpy(centroids, centroids_d, k * sizeof(Centroid), hipMemcpyDeviceToHost));

}

extern "C" void copyCentroidsToDevice(Centroid* deviceCentroids, Centroid* hostCentroids, int k) {
    checkCuda(hipMemcpy(deviceCentroids, hostCentroids, k * sizeof(Centroid), hipMemcpyHostToDevice));
// checkCuda(hipMemcpy(centroids_d, centroids, k * sizeof(Centroid), hipMemcpyHostToDevice));
}

extern "C" void copySongsToHost(Song* hostSongs, Song* deviceSongs, int localN) {
    checkCuda(hipMemcpy(hostSongs, deviceSongs, localN * sizeof(Song), hipMemcpyDeviceToHost));
// checkCuda(hipMemcpy(localSongs, localSongs_d, localN * sizeof(Song), hipMemcpyDeviceToHost));
}
