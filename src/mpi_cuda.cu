#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>
#include <random>
#include <sstream>
#include <fstream>
#include <assert.h>

struct Song {
    float feature1, feature2, feature3;
    int cluster;

    Song(): feature1(0.0), feature2(0.0), feature3(0.0), cluster(-1) {}

    Song(float f1, float f2, float f3):
        feature1(f1),
        feature2(f2),
        feature3(f3),
        cluster(-1)
    {}
};

struct Centroid {
    float feature1, feature2, feature3;
    int cluster_size;

    Centroid(): feature1(0.0), feature2(0.0), feature3(0.0), cluster_size(0) {}

    Centroid(float f1, float f2, float f3):
        feature1(f1),
        feature2(f2),
        feature3(f3),
        cluster_size(0)
    {}
};
inline hipError_t checkCuda(hipError_t result)
{
    if (result != hipSuccess) {
        std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
        assert(result == hipSuccess);
    }
    return result;
}
__device__ double sq_distance(Song* s1, Centroid* c)
{
    return (s1->feature1 - c->feature1) * (s1->feature1 - c->feature1) +
        (s1->feature2 - c->feature2) * (s1->feature2 - c->feature2) +
        (s1->feature3 - c->feature3) * (s1->feature3 - c->feature3);
}

__global__ void assignSongToCluster(Song* songs, Centroid* centroids, int n, int k)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    float minDist = 100000000;
    int cluster = -1;
    if (gid < n)
        for (int c = 0; c < k; ++c)
        {
            double dist = sq_distance(&songs[gid], &centroids[c]);
            if (dist < minDist)
            {
                minDist = dist;
                cluster = c;
            }
        }
    songs[gid].cluster = cluster;
}

__global__ void calculateNewCentroids(Song* songs, Centroid* centroids, int n)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < n)
    {
        int cluster = songs[gid].cluster; // Get the cluster of each song
        atomicAdd(&centroids[cluster].feature1, songs[gid].feature1);
        atomicAdd(&centroids[cluster].feature2, songs[gid].feature2);
        atomicAdd(&centroids[cluster].feature3, songs[gid].feature3);
        atomicAdd(&centroids[cluster].cluster_size, 1);
    }
}

// Wrapper function for assignSongToCluster kernel
extern "C" void callAssignSongToCluster(Song * songs, Centroid * centroids, int n, int k) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock; 

    dim3 grid(blocksPerGrid);
    dim3 block(threadsPerBlock);

    // Launch the kernel
    assignSongToCluster << <grid, block >> > (songs, centroids, n, k);

    // Check for errors and synchronize
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
    }
}

// Wrapper function for calculateNewCentroids kernel
extern "C" void callCalculateNewCentroids(Song * songs, Centroid * centroids, int n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    dim3 grid(blocksPerGrid);
    dim3 block(threadsPerBlock);

    // Launch the kernel
    calculateNewCentroids << <grid, block >> > (songs, centroids, n);

    // Check for errors and synchronize
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
    }
}

// Definitions of CUDA wrapper functions
extern "C" void allocateMemoryAndCopyToGPU(Song** deviceSongs, Centroid** deviceCentroids, const Song* hostSongs, const Centroid* hostCentroids, int numSongs, int numCentroids) {
    // Allocate memory for songs on the device
    checkCuda(hipMalloc(deviceSongs, numSongs * sizeof(Song)));
    checkCuda(hipMemcpy(*deviceSongs, hostSongs, numSongs * sizeof(Song), hipMemcpyHostToDevice));

    // Allocate memory for centroids on the device
    checkCuda(hipMalloc(deviceCentroids, numCentroids * sizeof(Centroid)));
    checkCuda(hipMemcpy(*deviceCentroids, hostCentroids, numCentroids * sizeof(Centroid), hipMemcpyHostToDevice));
}

extern "C" void freeGPUMemory(Song* deviceSongs, Centroid* deviceCentroids) {
    checkCuda(hipFree(deviceSongs));
    checkCuda(hipFree(deviceCentroids));
}

extern "C" void gpuErrorCheck() {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        assert(error == hipSuccess);
    }
    checkCuda(hipDeviceSynchronize());
}

extern "C" void resetCentroids(Centroid* centroids_d, int k) {
    checkCuda(hipMemset(centroids_d, 0, k * sizeof(Centroid)));
}

extern "C" void copyCentroidsToHost(Centroid* centroids, Centroid* centroids_d, int k) {
    checkCuda(hipMemcpy(centroids, centroids_d, k * sizeof(Centroid), hipMemcpyDeviceToHost));
}

extern "C" void copyCentroidsToDevice(Centroid* deviceCentroids, Centroid* hostCentroids, int k) {
    checkCuda(hipMemcpy(deviceCentroids, hostCentroids, k * sizeof(Centroid), hipMemcpyHostToDevice));
}

extern "C" void copySongsToHost(Song* hostSongs, Song* deviceSongs, int localN) {
    checkCuda(hipMemcpy(hostSongs, deviceSongs, localN * sizeof(Song), hipMemcpyDeviceToHost));
}
